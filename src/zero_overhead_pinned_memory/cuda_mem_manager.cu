#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <cstdlib>      // for posix_memalign, free
#include <cstring>      // for memset
#include <stdexcept>    // for std::runtime_error

#define CUDA_CHECK(call)                                                    \
  do {                                                                      \
    hipError_t err = call;                                                 \
    if (err != hipSuccess) {                                               \
      std::fprintf(stderr, "CUDA ERROR (%s:%d): %s\n",                      \
                   __FILE__, __LINE__, hipGetErrorString(err));            \
      std::exit(err);                                                       \
    }                                                                       \
  } while (0)

void posix_memory_free(void* ptr) {
    if (!ptr) {
        std::fprintf(stderr, "WARNING: Attempted to free a null pointer.\n");
        return;
    }

    // std::printf("INFO: Freeing pinned memory at address %p\n", ptr);

    CUDA_CHECK(hipHostUnregister(ptr)); 
    CUDA_CHECK(hipDeviceSynchronize());

    free(ptr);
}

torch::Tensor memalign_pin_memory(torch::Tensor& src) {
    TORCH_CHECK(src.device().is_cpu(), 
                "ERROR: Expected a CPU tensor, but received a tensor on device: ", 
                src.device().str());

    const size_t DMA_ALIGNMENT = 4096;
    size_t size = src.numel() * src.element_size(); 
    size_t allocate_size = ((size + DMA_ALIGNMENT - 1) / DMA_ALIGNMENT) * DMA_ALIGNMENT;

    void* data_ptr = nullptr;
    int ret = posix_memalign(&data_ptr, DMA_ALIGNMENT, allocate_size);
    if (ret != 0) {
        throw std::runtime_error("ERROR: Failed to allocate aligned memory. posix_memalign returned: " + std::to_string(ret));
    }

    std::memset(data_ptr, 0, allocate_size);

    CUDA_CHECK(hipHostRegister(data_ptr, allocate_size, hipHostRegisterDefault)); 
    CUDA_CHECK(hipDeviceSynchronize());

    auto tensor = torch::from_blob(data_ptr, src.sizes(), posix_memory_free, src.options());
    tensor.copy_(src);

    // std::printf("INFO: Allocated pinned memory at %p (size: %zu bytes, aligned to %zu bytes)\n", 
                // data_ptr, size, DMA_ALIGNMENT);

    return tensor;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
     m.def("memalign_pin_memory", &memalign_pin_memory,
          "Allocate pinned CPU memory aligned to 4096 bytes and copy the input tensor data into it. "
          "Returns a tensor with the same shape and options as the input.");
}